#include "hip/hip_runtime.h"
#include "utilities.cuh"

// CUDA kernel to compute denoise image
__global__ void nonLocalMeans(double* P, int m, int n, int w, double filtSigma, double* F) {

  __shared__ int size = (1024+(w-1))(1024+(w-1));
  __shared__ double localImageBlock[size];
  double Wxy = 0.0;
  double Zx = 0.0;
  double D = 0.0;

  int i = blockIdx.x;
  int j = threadIdx.x;

  F[i*n+j] = 0.0;
  Zx = 0.0;
  for(int k = 0; k < m; ++k) {
    for(int l = 0; l < n; ++l) {
        Wxy = 0.0;
        D = 0.0;
        for(int p = -(w-1)/2; p <= (w-1)/2; ++p) {
          for(int q = -(w-1)/2; q <= (w-1)/2; ++q) {
            // D += pow(P[(i+p)*(n+w-1)+(j+q)] - P[(k+p)*(n+w-1)+(l+q)], 2.0);
            D += pow((P[i*n*w*w+j*w*w+(p+(w-1)/2)*w+(q+(w-1)/2)] - P[k*n*w*w+l*w*w+(p+(w-1)/2)*w+(q+(w-1)/2)]), 2.0);

          }
        }
        Wxy = exp(-D/(filtSigma*filtSigma));
        Zx += Wxy;
        // P[k][l][(w-1)/2][(w-1)/2] is the center pixel of current patch
        F[i*n+j] += Wxy * P[k*n*w*w + l*w*w + (w-1)/2*w + (w-1)/2];
    }
  }
  F[i*n+j] /= Zx;

  return;
}

// Main function
int main(int argc, char* argv[]) {

  // Various checks for valid input arguments
  if(argc < 8) {
    printf("Usage: ./V1 m n w input_image output_image_name\n");
    return 1;
  }

  // Read input arguments
  int m = atoi(argv[1]);
  int n = atoi(argv[2]);
  int w = atoi(argv[3]);

  if(m != n) {
    printf("Only square images supported\n");
    return 1;
  }

  if(m != 64 && m != 128 && m != 256) {
    printf("Only 64x64, 128x128 and 256x256 size images supported\n");
    return 1;
  }

  if(w != 3 && w != 5 && w != 7) {
    printf("Only 3x3, 5x5 and 7x7 patch sizes supported\n");
    return 1;
  }

  double patchSigma = 2.0;
  sscanf(argv[4],"%lf",&patchSigma);

  double filtSigma = 0.02;
  sscanf(argv[5],"%lf",&filtSigma);

  // Create gaussian kernel
  double* W = (double*)malloc(w*w*sizeof(double));
  double sum = 0.0;
  for(int i = 0; i < w; ++i) {
    for(int j = 0; j < w; ++j) {
      W[i*w+j] = exp((-pow((double)(i-w/2)/(double)w, 2)-pow((double)(j-w/2)/(double)w, 2))/(2.0*patchSigma*patchSigma));
      sum += W[i*w+j];
    }
  }

  // Normalize
  for(int i = 0; i < w; ++i)
    for(int j = 0; j < w; ++j)
      W[i*w+j] /= sum;


  // Original Image extended to fit patches on the edges [(m+w-1)-by-(n+w-1)]
  double* X = (double*)malloc((m+w-1)*(n+w-1)*sizeof(double));
  // 3D Patch Cube [m-by-n-by-w-by-w]
  double* P = (double*)malloc(m*n*w*w*sizeof(double));
  // Filtered image [m-by-n]
  double* F = (double*)malloc(m*n*sizeof(double));
  // Residual image (F - x)
  double* R = (double*)malloc(m*n*sizeof(double));

  // 3D Patch cude pointer for GPU memory
  double* deviceP = NULL;
  hipMalloc(&deviceP, m*n*w*w*sizeof(double));

  // Filtered image pointer for GPU memory
  double* deviceF = NULL;
  hipMalloc(&deviceF, m*n*sizeof(double));

  FILE* fptr = fopen(argv[6], "r");
  for(int i = (w-1)/2; i < m+(w-1)/2; ++i)
    for(int j = (w-1)/2; j < n+(w-1)/2; ++j)
      fscanf(fptr, "%lf,", X+i*(n+w-1)+j);
      // X[i*(n+w-1)+j] = (i+j);
  fclose(fptr);

  // Add noise to input image
  for(int i = (w-1)/2; i < m+(w-1)/2; ++i)
    for(int j = (w-1)/2; j < n+(w-1)/2; ++j)
      X[i*(n+w-1)+j] += gaussianRand(0.04);


  // Fill edges mirroring the inside of image
  // similar to padarray(inputImage, [(w-1)/2 (w-1)/2], 'symmetric')

  // Right and left part
  for(int i = (w-1)/2; i < m+(w-1)/2; ++i) {

    for(int j = 0; j < (w-1)/2; ++j) {
      X[i*(n+w-1)+j] = X[i*(n+w-1)+(w-j-2)];
    }

    for(int j = 1; j <= (w-1)/2 ; ++j) {
      X[i*(n+w-1)+((n+w-1)-j)] = X[i*(n+w-1)+((n+w-1)-(w-(j-1)-1))];
    }

  }

  // Upper and lower part
  for(int i = 0; i < m+(w-1); ++i) {

    for(int j = 0; j < (w-1)/2; ++j) {
      X[j*(n+w-1)+i] = X[(w-j-2)*(n+w-1)+i];
    }

    for(int j = 1; j <= (w-1)/2 ; ++j) {
      X[((n+w-1)-j)*(n+w-1)+i] = X[((n+w-1)-(w-(j-1)-1))*(n+w-1)+i];
    }

  }

  // Calculate all w-by-w patches from X multiplied
  // with gaussian kernel and save them to P
  // (i,j) is the center pixel of each patch
  // (k,l) is the patch element
  // appropriate offsets are used
  for(int i = (w-1)/2; i < m+(w-1)/2;++i) {
      for(int j = (w-1)/2; j < n+(w-1)/2; ++j) {
        for(int k = -(w-1)/2; k <= (w-1)/2; ++k) {
            for(int l = -(w-1)/2; l <= (w-1)/2; ++l) {
              P[(i-(w-1)/2)*n*w*w+(j-(w-1)/2)*w*w+(k+(w-1)/2)*w+(l+(w-1)/2)] =
              X[(i+k)*(n+w-1)+(j+l)]*W[(k+(w-1)/2)*w+(l+(w-1)/2)];
            }
        }
      }
  }

  // Write noisy image to csv txt file
  // used by matlab script
  char outputFileName[100] = "";
  sprintf(outputFileName, "../output_images/output_images_csv_txt/output_images_V1/%s_%d_%d_noisy.txt", argv[7], n, w);
  // printf("Writing noisy image to %s\n", outputFileName);
  printMatrixCsv(X, m+w-1, n+w-1, outputFileName);

  // Copy data for input and output
  // from CPU memory to GPU memory
  hipMemcpy(deviceP, P, m*n*w*w*sizeof(double), hipMemcpyHostToDevice);
  hipMemcpy(deviceF, F, m*n*sizeof(double), hipMemcpyHostToDevice);

  // CUDA events used for measuring time
  hipEvent_t start, stop;
  hipEventCreate(&start);
  hipEventCreate(&stop);

  // Start measuring time and call kernel
  hipEventRecord(start);
  nonLocalMeans<<<m,n>>>(deviceP, m, n, w, filtSigma, deviceF);
  hipEventRecord(stop);

  // Copy data for input and output
  // from CPU memory to GPU memory
  hipMemcpy(P, deviceP, m*n*w*w*sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(F, deviceF, m*n*sizeof(double), hipMemcpyDeviceToHost);

  // Find original denoised pixel
  // divinding by center pixel
  // of gaussian kernel value
  for(int i = 0; i < m; ++i)
    for(int j = 0; j < n; ++j)
      F[i*n+j] /= W[(w-1)/2*w+(w-1)/2];

  hipEventSynchronize(stop);
  float milliseconds = 0;
  hipEventElapsedTime(&milliseconds, start, stop);

  // Calculate residual image
  for(int i = 0; i < m; ++i) {
    for(int j = 0; j < n; ++j) {
      R[i*n+j] = F[i*n+j] - X[(i+(w-1)/2)*(n+w-1)+(j+(w-1)/2)];
    }
  }

  // Write filtered image to csv txt file
  // used by matlab script
  sprintf(outputFileName, "../output_images/output_images_csv_txt/output_images_V1/%s_%d_%d_denoised.txt", argv[7], n, w);
  // printf("Writing denoised image to %s\n", outputFileName);
  printMatrixCsv(F, m, n, outputFileName);

  sprintf(outputFileName, "../output_images/output_images_csv_txt/output_images_V1/%s_%d_%d_residual.txt", argv[7], n, w);
  // printf("Writing residual image to %s\n", outputFileName);
  printMatrixCsv(R, m, n, outputFileName);

  printf("%lf\n", milliseconds);

  // Deallocate CPU and GPU memory
  hipFree(deviceP);
  hipFree(deviceF);
  free(X);
  free(F);
  free(W);
  free(P);
  free(R);

  return 0;
}
